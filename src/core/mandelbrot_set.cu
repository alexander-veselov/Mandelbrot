#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <stdexcept>

#include "mandelbrot_set.cuh"
#include "coloring.cuh"
#include "long_float.cuh"
#include "gpu_memory_pool.h"

namespace MandelbrotSet {

template <typename T>
__global__ void KernelMandelbrotSet(float_t* data, uint32_t width,
                                    uint32_t height, T center_real,
                                    T center_imag, T zoom_factor,
                                    uint32_t max_iterations,
                                    bool smoothing_step = false) {

  const auto pixel_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (pixel_index < width * height) {
    // Mandelbrot set parameters
    constexpr static auto kMandelbrotSetWidth  = 3.; // [-2, 1]
    constexpr static auto kMandelbrotSetHeight = 2.; // [-1, 1]

    const auto scale = fmin(width / kMandelbrotSetWidth, height / kMandelbrotSetHeight);

    const auto x = T{ (pixel_index % width - width  / 2.) / scale };
    const auto y = T{ (pixel_index / width - height / 2.) / scale };

    const auto real0 = center_real + x / zoom_factor;
    const auto imag0 = center_imag + y / zoom_factor;

    auto real = real0;
    auto imag = imag0;
    auto iterations = max_iterations;

    auto limit = 4.;
    if (smoothing_step) {
      limit = 16.;
    }

    for (auto i = uint32_t{0}; i < max_iterations; ++i) {
      const auto real_squared = real * real;
      const auto imag_squared = imag * imag;

      if (real_squared + imag_squared > limit) {
        iterations = i;
        break;
      }

      imag = T{2.} * real * imag + imag0;
      real = real_squared - imag_squared + real0;
    }

    if (smoothing_step && iterations < max_iterations) {
      const auto log_zn = log(static_cast<double_t>(real * real + imag * imag)) / 2.;
      const auto nu = static_cast<float_t>(log(log_zn / log(2.)) / log(2.));
      data[pixel_index] = static_cast<float_t>(iterations) + 1.f - nu;
    } else {
      data[pixel_index] = static_cast<float_t>(iterations);
    }
  }
}

void Visualize(uint32_t* image, uint32_t image_width, uint32_t image_height,
               double_t center_real, double_t center_imag, double_t zoom_factor,
               uint32_t max_iterations, uint32_t coloring_mode,
               bool smoothing) {

  static auto memory_pool = GPUMemoryPool{ 128 << 20 }; // 128 MB

  const auto image_size = image_width * image_height;
  const auto image_size_in_bytes = image_size * sizeof(uint32_t);

  auto device_data = memory_pool.Alloc(image_size_in_bytes);
  hipMemcpy(device_data, image, image_size_in_bytes, hipMemcpyHostToDevice);

  constexpr auto kThreadsPerBlock = 512;
  const auto kBlocksPerGrid = (image_size - 1) / kThreadsPerBlock + 1;

  using DataType = double_t;

  KernelMandelbrotSet<DataType><<<kBlocksPerGrid, kThreadsPerBlock>>>(
    reinterpret_cast<float_t*>(device_data), image_width, image_height,
    DataType{ center_real }, DataType{ center_imag }, DataType{ zoom_factor },
    max_iterations, smoothing);

  switch (coloring_mode) {
    case 1:
      Coloring::KenrelMode1<<<kBlocksPerGrid, kThreadsPerBlock>>>(
        reinterpret_cast<uint32_t*>(device_data), image_width,
        image_height, max_iterations);
      break;
    case 2:
      Coloring::KenrelMode2<<<kBlocksPerGrid, kThreadsPerBlock>>>(
        reinterpret_cast<uint32_t*>(device_data), image_width,
        image_height, max_iterations);
      break;
    case 3:
      Coloring::KenrelMode3<<<kBlocksPerGrid, kThreadsPerBlock>>>(
        reinterpret_cast<uint32_t*>(device_data), image_width,
        image_height, max_iterations);
      break;
    case 4:
      Coloring::KenrelMode4<<<kBlocksPerGrid, kThreadsPerBlock>>>(
        reinterpret_cast<uint32_t*>(device_data), image_width,
        image_height, max_iterations);
      break;
    case 5:
      Coloring::KenrelMode5<<<kBlocksPerGrid, kThreadsPerBlock>>>(
        reinterpret_cast<uint32_t*>(device_data), image_width,
        image_height, max_iterations);
      break;
    default:
      Coloring::KenrelDefaultMode<<<kBlocksPerGrid, kThreadsPerBlock>>>(
        reinterpret_cast<uint32_t*>(device_data), image_width,
        image_height, max_iterations);
  }

  hipMemcpy(image, device_data, image_size_in_bytes, hipMemcpyDeviceToHost);
  memory_pool.Free(device_data);
}

}  // namespace MandelbrotSet