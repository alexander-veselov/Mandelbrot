#include "hip/hip_runtime.h"
#include "mandelbrot/core/cuda/mandelbrot_set.h"

#include "mandelbrot/core/cuda/coloring.h"
#include "mandelbrot/core/gpu_memory_pool.h"

#include <hip/hip_runtime.h>
#include <>
#include <stdexcept>

namespace mandelbrot {
namespace cuda {

template <typename T>
__global__ void KernelMandelbrotSet(float_t* data, uint32_t width,
                                    uint32_t height, T center_real,
                                    T center_imag, T zoom_factor,
                                    uint32_t max_iterations,
                                    bool smoothing_step = false) {

  const auto pixel_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (pixel_index < width * height) {
    // Mandelbrot set parameters
    constexpr static auto kMandelbrotSetWidth  = T{3};  // [-2, 1]
    constexpr static auto kMandelbrotSetHeight = T{2};  // [-1, 1]

    const auto scale =
        T{1} / fmin(width / kMandelbrotSetWidth, height / kMandelbrotSetHeight);

    const auto x = (pixel_index % width - width  / T{2}) * scale;
    const auto y = (pixel_index / width - height / T{2}) * scale;

    const auto real0 = center_real + x / zoom_factor;
    const auto imag0 = center_imag + y / zoom_factor;

    auto real = real0;
    auto imag = imag0;
    auto iterations = max_iterations;

    auto limit = T{4};
    if (smoothing_step) {
      limit = T{16};
    }

    for (auto i = uint32_t{0}; i < max_iterations; ++i) {
      const auto real_squared = real * real;
      const auto imag_squared = imag * imag;

      if (real_squared + imag_squared > limit) {
        iterations = i;
        break;
      }

      imag = T{2} * real * imag + imag0;
      real = real_squared - imag_squared + real0;
    }

    if (smoothing_step && iterations < max_iterations) {
      const auto log_zn = logf(real * real + imag * imag) / 2.;
      const auto nu = static_cast<float_t>(logf(log_zn / logf(2)) / logf(2));
      data[pixel_index] = static_cast<float_t>(iterations) + 1.f - nu;
    } else {
      data[pixel_index] = static_cast<float_t>(iterations);
    }
  }
}

void Visualize(uint32_t* image, uint32_t image_width, uint32_t image_height,
               double_t center_real, double_t center_imag, double_t zoom_factor,
               uint32_t max_iterations, uint32_t coloring_mode,
               uint32_t palette, bool smoothing) {

  static auto memory_pool = GPUMemoryPool{ 128 << 20 }; // 128 MB

  const auto image_size = image_width * image_height;
  const auto image_size_in_bytes = image_size * sizeof(uint32_t);

  auto device_data = memory_pool.Alloc(image_size_in_bytes);
  hipMemcpy(device_data, image, image_size_in_bytes, hipMemcpyHostToDevice);

  constexpr auto kThreadsPerBlock = 512;
  const auto kBlocksPerGrid = (image_size - 1) / kThreadsPerBlock + 1;

  KernelMandelbrotSet<double_t><<<kBlocksPerGrid, kThreadsPerBlock>>>(
      reinterpret_cast<float_t*>(device_data), image_width, image_height,
      center_real, center_imag, zoom_factor, max_iterations, smoothing);

  cuda::KenrelColor<<<kBlocksPerGrid, kThreadsPerBlock>>>(
      reinterpret_cast<uint32_t*>(device_data), image_width, image_height,
      max_iterations, coloring_mode, palette);

  hipMemcpy(image, device_data, image_size_in_bytes, hipMemcpyDeviceToHost);
  memory_pool.Free(device_data);
}

}  // namespace cuda
}  // namespace mandelbrot