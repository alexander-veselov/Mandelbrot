#include "hip/hip_runtime.h"
#include "mandelbrot/core/cuda/coloring.h"

#include <hip/hip_runtime.h>

__device__ constexpr uint32_t MakeRGB(uint8_t r, uint8_t g, uint8_t b,
                                      uint8_t a = 255) {
  return r + (g << 8) + (b << 16) + (a << 24);
}

__device__ constexpr uint32_t InterpolateColor(uint32_t color1, uint32_t color2,
                                               double_t fraction) {

  const auto b1 = static_cast<uint8_t>((color1 >> 16) & 0xff);
  const auto b2 = static_cast<uint8_t>((color2 >> 16) & 0xff);
  const auto g1 = static_cast<uint8_t>((color1 >> 8) & 0xff);
  const auto g2 = static_cast<uint8_t>((color2 >> 8) & 0xff);
  const auto r1 = static_cast<uint8_t>(color1 & 0xff);
  const auto r2 = static_cast<uint8_t>(color2 & 0xff);

  const auto r = static_cast<uint32_t>((r2 - r1) * fraction + r1);
  const auto g = static_cast<uint32_t>((g2 - g1) * fraction + g1) << 8;
  const auto b = static_cast<uint32_t>((b2 - b1) * fraction + b1) << 16;
  constexpr auto kAlpha = static_cast<uint32_t>(uint8_t{255} << 24);

  return r + g + b + kAlpha;
}

__device__ void LchToRGB(double L, double C, double H, uint8_t& ROut, uint8_t& GOut,
                         uint8_t& BOut) {

  // Convert Lch to Lab
  auto A = C * cos(H * HIP_PI_F / 180.0);
  auto B = C * sin(H * HIP_PI_F / 180.0);

  // Convert Lab to XYZ
  auto Y = (L + 16.) / 116.;
  auto X = A / 500. + Y;
  auto Z = Y - B / 200.;

  if (pow(Y, 3) > 0.008856) {
    Y = pow(Y, 3);
  } else {
    Y = (Y - 16. / 116.) / 7.787;
  }

  if (pow(X, 3) > 0.008856) {
    X = pow(X, 3);
  }
  else {
    X = (X - 16. / 116.) / 7.787;
  }

  if (pow(Z, 3) > 0.008856) {
    Z = pow(Z, 3);
  } else {
    Z = (Z - 16. / 116.) / 7.787;
  }

  X *= 0.95047;
  Y *= 1.00000;
  Z *= 1.08883;

  // Convert XYZ to RGB
  auto R = X * +3.2406 + Y * -1.5372 + Z * -0.4986;
  auto G = X * -0.9689 + Y * +1.8758 + Z * +0.0415;
  /**/ B = X * +0.0557 + Y * -0.2040 + Z * +1.0570;

  if (R > 0.0031308) {
    R = 1.055 * pow(R, (1 / 2.4)) - 0.055;
  }
  else {
    R = 12.92 * R;
  }

  if (G > 0.0031308) {
    G = 1.055 * pow(G, (1 / 2.4)) - 0.055;
  }
  else {
    G = 12.92 * G;
  }

  if (B > 0.0031308) {
    B = 1.055 * pow(B, (1 / 2.4)) - 0.055;
  }
  else {
    B = 12.92 * B;
  }

  ROut = R * UINT8_MAX;
  GOut = G * UINT8_MAX;
  BOut = B * UINT8_MAX;
}

__device__ void HSVToRGB(double h, double s, double v, uint8_t& ROut,
                         uint8_t& GOut, uint8_t& BOut) {

  if (s <= 0.0) {
    ROut = v * UINT8_MAX;
    GOut = v * UINT8_MAX;
    BOut = v * UINT8_MAX;
    return;
  }

  auto hh = h;
  if (hh >= 360.0) {
    hh = 0.0;
  }
  hh /= 60.0;

  const auto i = static_cast<uint8_t>(hh);
  const auto ff = hh - i;
  const auto p = v * (1.0 - s);
  const auto q = v * (1.0 - (s * ff));
  const auto t = v * (1.0 - (s * (1.0 - ff)));

  auto R = double_t{};
  auto G = double_t{};
  auto B = double_t{};

  switch (i) {
    case 0:
      R = v;
      G = t;
      B = p;
      break;
    case 1:
      R = q;
      G = v;
      B = p;
      break;
    case 2:
      R = p;
      G = v;
      B = t;
      break;
    case 3:
      R = p;
      G = q;
      B = v;
      break;
    case 4:
      R = t;
      G = p;
      B = v;
      break;
    case 5:
    default:
      R = v;
      G = p;
      B = q;
      break;
  }

  ROut = R * UINT8_MAX;
  GOut = G * UINT8_MAX;
  BOut = B * UINT8_MAX;
}

namespace mandelbrot {
namespace cuda {

__device__ uint32_t DefaultMode(uint32_t iterations, uint32_t max_iterations) {

  const auto ratio = static_cast<double_t>(iterations) /
                     static_cast<double_t>(max_iterations);

  auto r = uint8_t{};
  auto g = uint8_t{};
  auto b = uint8_t{};

  r = g = b = ratio * UINT8_MAX;

  return MakeRGB(UINT8_MAX - r, UINT8_MAX - g, UINT8_MAX - b);
}

__device__ uint32_t Mode1(uint32_t iterations, uint32_t max_iterations) {

  static uint32_t palette[] = {
      MakeRGB(30,  16,  48), MakeRGB(15,  12,  94), MakeRGB(3,  28, 150),
      MakeRGB(6,  50, 188), MakeRGB(20,  80, 195), MakeRGB(57, 125, 209),
      MakeRGB(73, 155, 216), MakeRGB(134, 181, 229), MakeRGB(170, 210, 235),
      MakeRGB(240, 249, 250), MakeRGB(246, 231, 161), MakeRGB(252, 212,  70),
      MakeRGB(251, 175,  42), MakeRGB(215, 126,  41), MakeRGB(187,  79,  39),
      MakeRGB(110,  50,  53)
  };

  constexpr auto kPaletteSize = sizeof(palette) / sizeof(uint32_t);

  if (0 < iterations && iterations < max_iterations) {
    constexpr auto kStep = 64;
    constexpr auto kInterval = 1. / kPaletteSize;
    const auto hue = static_cast<double_t>(iterations % kStep) / kStep;
    const auto n = static_cast<uint32_t>(hue / kInterval);
    const auto fraction = fmod(hue, kInterval) / kInterval;
    return InterpolateColor(palette[n], palette[(n + 1) % kPaletteSize], fraction);
  }
  else {
    return MakeRGB(0, 0, 0);
  }
}

__device__ uint32_t Mode2(uint32_t iterations, uint32_t max_iterations) {

  const auto ratio = static_cast<double_t>(iterations) /
                     static_cast<double_t>(max_iterations);

  const auto h = fmod(pow(ratio * 360., 1.5), 360.);
  const auto s = 100.;
  const auto v = ratio * 100.;

  auto r = uint8_t{};
  auto g = uint8_t{};
  auto b = uint8_t{};

  HSVToRGB(h, s, v, r, g, b);

  return MakeRGB(r, g, b);
}

__device__ uint32_t Mode3(uint32_t iterations, uint32_t max_iterations) {

  const auto ratio = static_cast<double_t>(iterations) /
                     static_cast<double_t>(max_iterations);

  const auto t = ratio;
  const auto r = static_cast<uint8_t>(9. * (1 - t) * t * t * t * UINT8_MAX);
  const auto g = static_cast<uint8_t>(15. * (1 - t) * (1 - t) * t * t * UINT8_MAX);
  const auto b = static_cast<uint8_t>(8.5 * (1 - t) * (1 - t) * (1 - t) * t * UINT8_MAX);

  return MakeRGB(r, g, b);
}

__device__ uint32_t Mode4(uint32_t iterations, uint32_t max_iterations) {

  const auto ratio = static_cast<double_t>(iterations) /
                     static_cast<double_t>(max_iterations);

  const auto hue = 240. * sqrt(ratio);

  const auto c = 1.;
  const auto x = c * (1. - fabs(fmod(hue / 60., 2.) - 1.));
  const auto m = 0.;

  auto r = double_t{};
  auto g = double_t{};
  auto b = double_t{};

  if (hue < 60.) {
    r = c;
    g = x;
    b = 0;
  } else if (hue < 120.) {
    r = x;
    g = c;
    b = 0;
  } else if (hue < 180.) {
    r = 0;
    g = c;
    b = x;
  } else if (hue < 240.) {
    r = 0;
    g = x;
    b = c;
  } else if (hue < 300.) {
    r = x;
    g = 0;
    b = c;
  } else {
    r = c;
    g = 0;
    b = x;
  }

  r = (r + m) * UINT8_MAX;
  g = (g + m) * UINT8_MAX;
  b = (b + m) * UINT8_MAX;

  return MakeRGB(r, g, b);
}

__device__ uint32_t Mode5(uint32_t iterations, uint32_t max_iterations) {

  const auto ratio = static_cast<double_t>(iterations) /
    static_cast<double_t>(max_iterations);

  const auto v = 1.0 - pow(cos(HIP_PI_F * ratio), 2.);
  const auto a = 111.;
  const auto L = a - (a * v);
  const auto C = 28. + (a - (a * v));
  const auto H = fmod(pow(360. * ratio, 1.5), 360.);

  auto r = uint8_t{};
  auto g = uint8_t{};
  auto b = uint8_t{};

  LchToRGB(L, C, H, r, g, b);

  return MakeRGB(r, g, b);
}

template <typename ColoringFunction>
__device__ void SmoothColor(ColoringFunction coloring_function, uint32_t* data,
                            uint32_t image_width, uint32_t image_height,
                            uint32_t max_iterations) {

  const auto pixel_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (pixel_index < image_width * image_height) {
    const auto iterations_f = reinterpret_cast<float_t*>(data)[pixel_index];
    const auto iterations = static_cast<uint32_t>(floorf(iterations_f));
    if (iterations >= max_iterations) {
      data[pixel_index] = MakeRGB(0, 0, 0);
      return;
    }

    const auto color1 = coloring_function(iterations, max_iterations);
    const auto color2 = coloring_function(iterations + 1, max_iterations);
    const auto fraction = fmod(iterations_f, 1.f);

    data[pixel_index] = InterpolateColor(color1, color2, fraction);
  }
}

template <typename ColoringFunction>
__device__ void NativeColor(ColoringFunction coloring_function, uint32_t* data,
                            uint32_t image_width, uint32_t image_height,
                            uint32_t max_iterations) {
  const auto pixel_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (pixel_index < image_width * image_height) {
    const auto iterations_f = reinterpret_cast<float_t*>(data)[pixel_index];
    const auto iterations = static_cast<uint32_t>(floorf(iterations_f));
    if (iterations >= max_iterations) {
      data[pixel_index] = MakeRGB(0, 0, 0);
      return;
    }

    data[pixel_index] = coloring_function(iterations, max_iterations);
  }
}

__global__ void KenrelDefaultMode(uint32_t* data, uint32_t image_width,
                                  uint32_t image_height,
                                  uint32_t max_iterations) {
  SmoothColor(DefaultMode, data, image_width, image_height, max_iterations);
}

__global__ void KenrelMode1(uint32_t* data, uint32_t image_width,
                            uint32_t image_height, uint32_t max_iterations) {
  SmoothColor(Mode1, data, image_width, image_height, max_iterations);
}

__global__ void KenrelMode2(uint32_t* data, uint32_t image_width,
                            uint32_t image_height, uint32_t max_iterations) {

  SmoothColor(Mode2, data, image_width, image_height, max_iterations);
}

__global__ void KenrelMode3(uint32_t* data, uint32_t image_width,
                            uint32_t image_height, uint32_t max_iterations) {
  SmoothColor(Mode3, data, image_width, image_height, max_iterations);
}

__global__ void KenrelMode4(uint32_t* data, uint32_t image_width,
                            uint32_t image_height, uint32_t max_iterations) {
  SmoothColor(Mode4, data, image_width, image_height, max_iterations);
}

__global__ void KenrelMode5(uint32_t* data, uint32_t image_width,
                            uint32_t image_height, uint32_t max_iterations) {
  SmoothColor(Mode5, data, image_width, image_height, max_iterations);
}

}  // namespace cuda
}  // namespace mandelbrot