#include <hip/hip_runtime.h>


#include <cmath>
#include <cstdint>

__device__ constexpr uint32_t MakeRGB(uint8_t r, uint8_t g, uint8_t b) {
  return b + (g << 8) + (r << 16);
}

template <typename DataType>
__device__ void ColorPixel(DataType* data, uint32_t pixel_index,
                           uint32_t iterations, uint32_t max_iterations) {
  if constexpr (sizeof(DataType) == sizeof(uint8_t)) {
    data[pixel_index] = iterations * UINT8_MAX / max_iterations;
  }

  if constexpr (sizeof(DataType) == sizeof(uint32_t)) {
    // TODO: fix code-style
    double r, g, b;
    double r1 = 0, b1 = max_iterations / 16, gr = max_iterations / 8,
           r2 = max_iterations / 4, b2 = max_iterations / 2;

    if (iterations < 0) {
      data[pixel_index] = MakeRGB(0, 0, 0);
    }

    if (iterations == 0) {
      r = max_iterations;
      g = 0;
      b = 0;
    } else {
      if (iterations < b1) {
        r = b1 * (b1 - iterations);
        g = 0;
        b = b1 * iterations - 1;
      } else if (iterations < gr) {
        r = 0;
        g = b1 * (iterations - b1);
        b = b1 * (gr - iterations) - 1;
      } else if (iterations < r2) {
        r = 8 * (iterations - gr);
        g = 8 * (r2 - iterations) - 1;
        b = 0;
      } else {
        r = max_iterations - (iterations - r2) * 4;
        g = 0;
        b = 0;
      }
    }
    data[pixel_index] = MakeRGB(r, g, b);
  }
}

template <typename DataType, typename T>
__global__ void MandelbrotSet(DataType* data, uint32_t width, uint32_t height,
                              T center_real, T center_imag, T zoom_factor) {

  const auto pixel_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (pixel_index < width * height) {
    constexpr static auto kMaxIterations = 1024;

    // Mandelbrot set parameters
    constexpr static auto kMandelbrotSetWidth  = T{3};  // [-2, 1]
    constexpr static auto kMandelbrotSetHeight = T{2};  // [-1, 1]

    const auto scale =
        T{1} / fmin(width / kMandelbrotSetWidth, height / kMandelbrotSetHeight);

    const auto x = (pixel_index % width - width  / T{2}) * scale;
    const auto y = (pixel_index / width - height / T{2}) * scale;

    const auto real0 = center_real + x / zoom_factor;
    const auto imag0 = center_imag + y / zoom_factor;

    auto real = real0;
    auto imag = imag0;

    for (auto i = uint32_t{0}; i < kMaxIterations; ++i) {
      const auto r2 = real * real;
      const auto i2 = imag * imag;

      if (r2 + i2 > T{4}) {
        ColorPixel(data, pixel_index, i, kMaxIterations);
        return;
      }

      imag = T{2} * real * imag + imag0;
      real = r2 - i2 + real0;
    }

    ColorPixel(data, pixel_index, kMaxIterations, kMaxIterations);
  }
}

template <typename T>
void MandelbrotSetImpl(T* image, uint32_t image_width, uint32_t image_height,
                       double_t center_real, double_t center_imag,
                       double_t zoom_factor) {
  const auto image_size = image_width * image_height;
  const auto image_size_in_bytes = image_size * sizeof(T);

  auto device_data = static_cast<T*>(nullptr);
  hipMalloc(&device_data, image_size_in_bytes);
  hipMemcpy(device_data, image, image_size_in_bytes, hipMemcpyHostToDevice);

  constexpr auto kThreadsPerBlock = 512;
  const auto kBlocksPerGrid = (image_size - 1) / kThreadsPerBlock + 1;

  MandelbrotSet<T, double_t><<<kBlocksPerGrid, kThreadsPerBlock>>>(
      device_data, image_width, image_height, center_real, center_imag,
      zoom_factor);

  hipMemcpy(image, device_data, image_size_in_bytes, hipMemcpyDeviceToHost);
}

void MandelbrotSet(uint8_t* image, uint32_t image_width, uint32_t image_height,
                   double_t center_real, double_t center_imag,
                   double_t zoom_factor) {
  MandelbrotSetImpl(image, image_width, image_height, center_real, center_imag,
                    zoom_factor);
}

void MandelbrotSet(uint32_t* image, uint32_t image_width, uint32_t image_height,
                   double_t center_real, double_t center_imag,
                   double_t zoom_factor) {
  MandelbrotSetImpl(image, image_width, image_height, center_real, center_imag,
                    zoom_factor);
}