#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>

#include "mandelbrot_set.cuh"

using ImageType = uint8_t;
using Image = std::vector<ImageType>;

constexpr auto kEnableVSync = false;
constexpr auto kWindowWidth = 1024;
constexpr auto kWindowHeight = 768;
constexpr auto kSize = kWindowWidth * kWindowHeight;
constexpr auto kSizeInBytes = kSize * sizeof(ImageType);

class FPSCounter {
 public:
  explicit FPSCounter(int32_t update_rate, double_t current_time)
      : update_rate_{update_rate},
        frame_count_{0},
        last_capture_time_{current_time},
        fps_{0.} {}

  void Update(double current_time) {
    const auto delta_time = current_time - last_capture_time_;
    fps_ = static_cast<double_t>(++frame_count_) / delta_time;
    if (delta_time * update_rate_ >= 1.) {
      last_capture_time_ = current_time;
      frame_count_ = 0;
    }
  }

  double_t GetFPS() const noexcept { return fps_; }

 private:
  const int32_t update_rate_;
  int32_t frame_count_;
  double_t last_capture_time_;
  double_t fps_;
};

void DrawImage(const Image& image) {
  glBegin(GL_POINTS);
  for (auto y = 0; y < kWindowHeight; ++y) {
    for (auto x = 0; x < kWindowWidth; ++x) {
      const auto pixel = image[y * kWindowWidth + x];
      glColor3ub(pixel, pixel, pixel);
      glVertex2d(x, y);
    }
  }
  glEnd();
}

void MandelbrotSet(Image& image) {
  auto device_data = static_cast<uint8_t*>(nullptr);
  hipMalloc(&device_data, kSizeInBytes);
  hipMemcpy(device_data, image.data(), kSizeInBytes, hipMemcpyHostToDevice);

  constexpr auto kThreadsPerBlock = 512;
  constexpr auto kBlocksPerGrid = (kSize - 1) / kThreadsPerBlock + 1;

  constexpr static auto kDx = -0.87591;
  constexpr static auto kDy = 0.20464;
  constexpr static auto kZoom = 5.7854;

  MandelbrotSet<<<kBlocksPerGrid, kThreadsPerBlock>>>(
      device_data, kWindowWidth, kWindowHeight, kDx, kDy, kZoom
  );

  hipMemcpy(image.data(), device_data, kSizeInBytes, hipMemcpyDeviceToHost);
}

int main() {

  if (glfwInit() != GLFW_TRUE) {
    glfwTerminate();
  }

  const auto window = glfwCreateWindow(kWindowWidth, kWindowHeight, "Mandelbrot set", nullptr, nullptr);

  glfwMakeContextCurrent(window);
  glfwSwapInterval(kEnableVSync);

  glViewport(0, 0, kWindowWidth, kWindowHeight);
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();

  glOrtho(0.0, kWindowWidth, 0.0, kWindowHeight, 0.0, 1.0);

  constexpr auto kFPSUpdateRate = 10; // 10 times per second
  const auto current_time = glfwGetTime();
  auto fps_counter = FPSCounter{kFPSUpdateRate, current_time};

  auto image = std::vector<uint8_t>(kSize);

  while (!glfwWindowShouldClose(window)) {

    MandelbrotSet(image);
    DrawImage(image);

    glfwSwapBuffers(window);
    glfwPollEvents();

    fps_counter.Update(glfwGetTime());
    std::cout << fps_counter.GetFPS() << std::endl;
  }

  glfwDestroyWindow(window);
  glfwTerminate();

  return 0;
}